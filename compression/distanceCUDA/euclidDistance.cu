#include "hip/hip_runtime.h"
#include "euclidDistance.h"
#include <float.h>
#include <cmath>
#include <hipcub/hipcub.hpp>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

/**************************************************SHARED********************************************************************/

__host__ __device__ float euclidDistance(const int dimSize, const float *data, const float *cluster) {
  float result = 0.0;
  for (int i = 0; i < dimSize; i++) {
    float diff = data[i] - cluster[i];
    result += diff * diff;
  }

  return result;
}

// Multiple atomic swap solution:
// https://stackoverflow.com/questions/17411493/how-can-i-implement-a-custom-atomic-function-involving-several-variables 
typedef union {
  float floats[2];                 // floats[0] = lowest
  int ints[2];                     // ints[1] = lowIdx
  unsigned long long int ulong;    // for atomic update
} myAtomics;

__device__ unsigned long long int myAtomicMin(unsigned long long int* address, float val1, int val2) {
    myAtomics loc, loctest;
    loc.floats[0] = val1;
    loc.ints[1] = val2;
    loctest.ulong = *address;
    while (loctest.floats[0] >  val1) { // Data can only get smaller -> no race condition possible
      loctest.ulong = atomicCAS(address, loctest.ulong, loc.ulong); // Reassign in case of race condition, if success assignment loc.ulong -> extra iteration needed to break out of loop
    }
    return loctest.ulong;
}

/**************************************************KERNELS********************************************************************/

__global__ void euclidDistanceIndexed(
  int dataSize, int clusterSize, int dimSize,
  const float *data,
  const float *clusters,
  int64_t *indices
) {

  auto idx = cg::this_grid().thread_rank();
  if (idx >= dataSize)
    return;

  float minDistance = FLT_MAX;
  int64_t minIndex;
  for (int64_t i = 0; i < clusterSize; i++) {
    float d = euclidDistance(dimSize, &(data[idx * dimSize]), &(clusters[i * dimSize]));
    if (d < minDistance) {
      minDistance = d; minIndex = i;
    }
  }

  indices[idx] = minIndex;
}

__global__ void euclidDistanceMapped(
  int dataSize, int dimSize,
  int sampleSetOneId,
  const float *sampleSetOne,
  const float *sampleSetTwo,
  int *sampleMapOne,
  int *sampleMapTwo,
  int *firstSampleMap,
  int *verticesMapped
) {

  auto idx = cg::this_grid().thread_rank();
  // Vertice is not mapped
  if (idx >= dataSize || sampleMapOne[idx * 2 + 1] == -2)
    return;

  float minDistance = FLT_MAX;
  int64_t minIndex;
  for (int64_t i = 0; i < dataSize; i++) {
    float d = euclidDistance(dimSize, &(sampleSetOne[idx * dimSize]), &(sampleSetTwo[i * dimSize]));
    if (d < minDistance) {
      minDistance = d; minIndex = i;
    }
  }

  // Exchange tail first to avoid race conditions - see initSampleSets for "if" explanation
  auto old = atomicExch(&(sampleMapTwo[minIndex * 2 + 1]), sampleMapOne[idx * 2 + 1]);
  if (old == -2) {
    sampleMapTwo[minIndex * 2] = dataSize * sampleSetOneId + idx;
    atomicAdd(verticesMapped, 1);
  }
  else {
    firstSampleMap[old * 2] = dataSize * sampleSetOneId + idx;
  }
}

__global__ void euclidDistanceMinimal(
  int dataSize, int dimSize,
  const float *data,
  myAtomics *minPoint
) {

  auto idx = cg::this_grid().thread_rank();
  if (idx >= dataSize)
    return;

  float totalDistance = 0.0;
  for (int64_t i = 0; i < dataSize; i++) {
    totalDistance += euclidDistance(dimSize, &(data[idx * dimSize]), &(data[i * dimSize]));;
  }

  myAtomicMin(&(minPoint->ulong), totalDistance, (int)idx);
}

/**************************************************C++********************************************************************/

torch::Tensor euclidDistanceIndexedCUDA(const torch::Tensor &data, const torch::Tensor &clusters) {
  if (data.ndimension() != 2 || clusters.ndimension() != 2) {
    AT_ERROR("Data and clusters must have dimension 2");
  }

  if (data.size(1) != clusters.size(1)) {
    AT_ERROR("Data and clusters must have same number of channels");
  }

  torch::Tensor indices = torch::empty({data.size(0)}, data.options().dtype(torch::kInt64));
  int dataSize = data.size(0);
  int dimSize = data.size(1);
  int clusterSize = clusters.size(0);

  // +31 for rounding
  euclidDistanceIndexed<<<(dataSize + 31) / 32, 32>>>(
      dataSize, clusterSize, dimSize,
      data.contiguous().data_ptr<float>(),
      clusters.contiguous().data_ptr<float>(),
      indices.contiguous().data_ptr<int64_t>()
  );

  return indices;
}

int euclidDistanceMappedCUDA(
    int sampleSetOneId,
    const torch::Tensor& sampleSetOne,
    const torch::Tensor& sampleSetTwo,
    torch::Tensor& sampleMapOne,
    torch::Tensor& sampleMapTwo,
    torch::Tensor& firstSampleMap
) {
  if (sampleSetOne.ndimension() != 2 || sampleSetTwo.ndimension() != 2) {
    AT_ERROR("Sample sets must have dimension 2");
  }

  if (sampleSetOne.size(0) != sampleSetTwo.size(0)) {
    AT_ERROR("Sample sets must have the same cardinality");
  }

  if (sampleSetOne.size(1) != sampleSetTwo.size(1)) {
    AT_ERROR("Sample sets must have the same number of channels");
  }

  int *verticesMapped;
  int zero = 0;
  hipMalloc(&verticesMapped, sizeof(int));
  hipMemcpy(verticesMapped, &zero, sizeof(int), hipMemcpyHostToDevice);
  int dataSize = sampleSetOne.size(0);
  int dimSize = sampleSetOne.size(1);

  // +31 for rounding
  euclidDistanceMapped<<<(dataSize + 31) / 32, 32>>>(
      dataSize, dimSize,
      sampleSetOneId,
      sampleSetOne.contiguous().data_ptr<float>(),
      sampleSetTwo.contiguous().data_ptr<float>(),
      sampleMapOne.contiguous().data_ptr<int>(),
      sampleMapTwo.contiguous().data_ptr<int>(),
      firstSampleMap.contiguous().data_ptr<int>(),
      verticesMapped
  );
  hipDeviceSynchronize();


  int copy;
  hipMemcpy(&copy, verticesMapped, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(verticesMapped);

  return copy;
}

std::vector<torch::Tensor> collectGroups(
    int sampleSize,
    int numClusters,
    const torch::Tensor& sampleSets,
    const torch::Tensor& sampleMaps
) {
  std::vector<torch::Tensor> ret{};
  std::vector<torch::Tensor> val{};

  int numSampleSets = sampleSets.size(0);
  for (int i = 0; i < sampleMaps.size(1); ++i) {
    int next = sampleMaps[numSampleSets - 1][i][0].item<int>();
    if (next == -2) {
      continue;
    }
    val.push_back(sampleSets[numSampleSets - 1][i]);
    while (next != -1) {
      int setId = floor(next / sampleSize);
      int offset = next % sampleSize;
      val.push_back(sampleSets[setId][offset]);
      next = sampleMaps[setId][offset][0].item<int>();
    }

    ret.push_back(torch::stack(val));
    val.clear();

    // Because of margin, take first n clusters
    if (ret.size() == numClusters) {
      return ret;
    }
  }

  return ret;
}

int findMedoids(
    const torch::Tensor& cluster
) {
  myAtomics *minPoint;
  float max = FLT_MAX;
  hipMalloc(&minPoint, sizeof(myAtomics));
  hipMemcpy(&(minPoint->floats[0]), &max, sizeof(myAtomics), hipMemcpyHostToDevice);

  int dataSize = cluster.size(0);
  int dimSize = cluster.size(1);

  // +31 for rounding
  euclidDistanceMinimal<<<(dataSize + 31) / 32, 32>>>(
      dataSize, dimSize,
      cluster.contiguous().data_ptr<float>(),
      minPoint
  );
  hipDeviceSynchronize();

  int copy;
  hipMemcpy(&copy, &(minPoint->ints[1]), sizeof(int), hipMemcpyDeviceToHost);
  hipFree(minPoint);

  return copy;

}

// Modified from "Compressed 3D Gaussian Splatting for Accelerated Novel View Synthesis":
// https://github.com/KeKsBoTer/c3dgs/blob/master/submodules/weighted_distance/weighted_distance.cu